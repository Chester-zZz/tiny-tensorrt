#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <plugin/PRELUKernel.cuh>

// CUDA: use 512 threads per block
const int CAFFE_CUDA_NUM_THREADS = 512;

// CUDA: number of blocks for threads.
inline int CAFFE_GET_BLOCKS(const int N) {
  return (N + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
}

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

/******** PReLU CUDA function ********/
// CUDA kernele for forward
__global__ void PReLUForward(const int n, const int channels, const int dim,
    const float* slope_data,
    const float* in, float* out,
    const float zero,
    const int div_factor) {
    CUDA_KERNEL_LOOP(index, n) {
        int c = (index / dim) % channels / div_factor;
        out[index] = in[index] > 0 ? in[index] : in[index] * slope_data[c];
    }
}


hipError_t Forward_gpu(const int count, const int channels, const int dim,
                const float* mDeviceKernel,
                const float* bottom_data, float* top_data, 
                const float zero,
                const int div_factor, const hipStream_t stream) {
    PReLUForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>
        (count, channels, dim, mDeviceKernel, bottom_data, top_data, zero, div_factor);
    hipError_t err = hipGetLastError();
    return err;
}